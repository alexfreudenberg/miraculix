/*
 Alexander Freudenberg, alexander.freudenberg@stads.de

 Copyright (C) 2020-2023 Alexander Freudenberg

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include <inttypes.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"


extern "C" {
int cublas_uint8_gemm(unsigned char *snp_matrix, int snps, int indiv,
                       double *ans) {
    /*
    xx
    */
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    hipError_t err = hipSuccess;
    hipblasHandle_t handle;

    hipblasGemmAlgo_t    algo         = HIPBLAS_GEMM_DEFAULT;
    hipDataType      input_type   = HIP_R_8I;
    hipDataType      output_type  = HIP_R_32F;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    void  *d_A = NULL,
          *d_B = NULL;
    float *d_C = NULL,
          *h_C = NULL;

    int nrowA = indiv,
        ncolA = snps;

    const float alpha = 1.0,
          beta        = 0.0;
    
    size_t size_of_input  = sizeof(uint8_t) * snps * indiv;
    size_t size_of_output = sizeof(float) * indiv * indiv;
    // Create handle
    hipblasCreate(&handle);

    // Allocate memory
    err = hipMalloc(&d_A, size_of_input);  
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);;
    err = hipMalloc(&d_B, size_of_input);  
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);;
    err = hipMalloc((void**)&d_C, size_of_output);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    err = hipHostMalloc((void**)&h_C, size_of_output);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);

    // Copy data to device
    err = hipMemcpy(d_A, snp_matrix, size_of_input, hipMemcpyHostToDevice);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    err = hipMemcpy(d_B, snp_matrix, size_of_input, hipMemcpyHostToDevice);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    
    // Calculate GEMM
    status =
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nrowA, nrowA, ncolA,
                     &alpha, d_A, input_type, nrowA, d_B, input_type, nrowA,
                     &beta, d_C, output_type, nrowA, compute_type, algo);
    hipDeviceSynchronize();
    if (checkError(__func__, __LINE__, status) != 0)
        return (1);

    // Copy data back to host
    err = hipMemcpy(h_C, d_C, size_of_output, hipMemcpyDeviceToHost);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);

    // Cast to double
    for(int i = 0; i < indiv * indiv; i++){
        ans[i] = (double) (h_C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_C);
    hipblasDestroy(handle);
    return 0;
}

}