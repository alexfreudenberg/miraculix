/*
 Alexander Freudenberg, alexander.freudenberg@stads.de

 Copyright (C) 2020-2023 Alexander Freudenberg

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include <inttypes.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"


extern "C" {
int cublas_uint8_gemm(unsigned char *snp_matrix, int snps, int indiv,
                       double *ans) {
    /*
    xx
    */
    hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;
    hipError_t err = hipSuccess;
    hipblasHandle_t handle;

    hipblasGemmAlgo_t    algo         = HIPBLAS_GEMM_DEFAULT;
    hipDataType      input_type   = HIP_R_8I;
    hipDataType      output_type  = HIP_R_32F;
    hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_32F;

    void  *d_A = NULL,
          *d_B = NULL;
    float *d_C = NULL,
          *h_C = NULL;

    size_t nrowA = ((indiv - 1)/4 + 1) * 4,
           ncolA = ((snps - 1)/4 + 1) * 4;
    debug_info("cuBLAS uint8: Problem size: (%ld, %ld).\n", nrowA, ncolA);

    const float alpha = 1.0,
          beta        = 0.0;
    
    size_t size_of_input  = sizeof(uint8_t) * nrowA * ncolA;
    size_t size_of_output = sizeof(float) * nrowA * nrowA;

    if (checkDevMemory(2 * size_of_input + size_of_output) != 0) {
      return 1;
    }
    // Create handle
    hipblasCreate(&handle);

    // Allocate memory
    err = hipMalloc(&d_A, size_of_input);  
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);;
    err = hipMalloc(&d_B, size_of_input);  
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);;
    err = hipMalloc((void**)&d_C, size_of_output);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    err = hipHostMalloc((void**)&h_C, size_of_output);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);

    // Copy data to device
    err = hipMemcpy2D(d_A, sizeof(unsigned char) * nrowA, snp_matrix,
                       sizeof(unsigned char) * indiv,
                       sizeof(unsigned char) * indiv,
                       sizeof(unsigned char) * snps, hipMemcpyHostToDevice);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    err = hipMemcpy2D(d_B, sizeof(unsigned char) * nrowA, snp_matrix,
                       sizeof(unsigned char) * indiv,
                       sizeof(unsigned char) * indiv,
                       sizeof(unsigned char) * snps, hipMemcpyHostToDevice);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    hipDeviceSynchronize();
    if (checkError(__func__, __LINE__, hipGetLastError()) != 0)
        return (1);

    // Calculate GEMM
    status =
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, nrowA, nrowA, ncolA,
                     &alpha, d_A, input_type, nrowA, d_B, input_type, nrowA,
                     &beta, d_C, output_type, nrowA, compute_type, algo);
    hipDeviceSynchronize();
    if (checkError(__func__, __LINE__, status) != 0)
        return (1);
    if (checkError(__func__, __LINE__, hipGetLastError()) != 0)
        return (1);

    // Copy data back to host
    err = hipMemcpy(h_C, d_C, size_of_output, hipMemcpyDeviceToHost);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    hipDeviceSynchronize();

    // Cast to double
    for (long i = 0; i < indiv; i++) {
        for(long j = 0; j < indiv; j++){
            ans[j + i * indiv] = (double)(h_C[j + i * nrowA]);
        }
    }
    if (checkError(__func__, __LINE__, hipGetLastError()) != 0)
        return (1);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_C);
    hipblasDestroy(handle);
    return 0;
}

}