#include "hip/hip_runtime.h"

/* 
 Authors
 Martin Schlather, martin.schlather@uni-mannheim.de

 (library for simulation of random fields)

 Copyright (C) 2021 -- 2021 Alexander Freudenberg

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 3
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.
*/


#define checkCudaErrors(err) {					      \
    hipError_t e=hipGetLastError();                                 \
    if(e!=hipSuccess) {                                              \
      PRINTF("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
      exit(0);								\
    }									\
  }

#include <hipsolver.h>
//#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <chrono>

#include "Basic_miraculix.h"
#include "xport_import.h"
#include "options.h"
#include "solve_gpu.h"
//#include "MXinfo.h"
//#include "xport_import.h"

//Two small kernels for printing data on the GPU, shoul be called with 1 block and 1 thread
/*__global__ void print_kernel(int32_t* d_C, int in dividuals, int snps) {
  for (int i = 0; i < in dividuals * snps;i++) p rintf("% " PRId32, d_C[i] );
}

__global__ void print_kernel(int8_t* d_C, int in dividuals, int snps) {
    for (int i = 0; i < in dividuals * snps;p rintf( (++i % individuals) ? "" : "\n")   ) p rintf("% " PRId8, d_C[i] );
}
*/

void gpuSolve(double *matrix, Uint individuals, double *vector){
/*
    This function solves the problem
        A x = b
    on an available GPU and writes the solution to the original memory
    Input: 
        matrix: pointer to rowwise allocated matrix A
        individuals: number of individuals in matrix, i.e. dimension
        vector: pointer to vector b
    Ouput:
        vector: contains solution x after the function has been called
*/

//declare/define process variables


  
    int bufferSize = 0;
    int *info = NULL;
    int h_info = 0;
    double *buffer = NULL;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipsolverHandle_t handle = NULL;
    hipStream_t stream = NULL;

//declare device variables
    Uint ind_sq = individuals * individuals;
    double *d_matrix = NULL;
    double *d_b = NULL; 

//initialize handle and stream, calculate buffer size needed for cholesky
    checkCudaErrors(hipsolverDnCreate(&handle));

    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    checkCudaErrors(hipsolverSetStream(handle, stream));

    checkCudaErrors(hipsolverDnDpotrf_bufferSize(handle, uplo, individuals, matrix,
        individuals, &bufferSize));
    checkCudaErrors(hipMalloc(&info, sizeof(int)));
    checkCudaErrors(hipMalloc(&buffer, sizeof(double) * bufferSize));
//allocate memory on device  
    checkCudaErrors(hipMalloc((void**)&d_matrix, sizeof(double)*ind_sq));
    checkCudaErrors(hipMalloc((void **)&d_b, sizeof(double) * individuals));
    checkCudaErrors(hipMemset(info, 0, sizeof(int)));

//coppy data to device
    checkCudaErrors(hipMemcpy(d_matrix, matrix, sizeof(double)*ind_sq, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b, vector, sizeof(double) * individuals, hipMemcpyHostToDevice));
    checkCudaErrors(hipDeviceSynchronize());

//write cholesky factorization to device copy of A
    hipsolverDnDpotrf(handle, uplo, individuals,
            d_matrix, individuals, buffer, bufferSize, info);
            
    //Synchronize is necessary, otherwise error code "info" returns nonsense 
    checkCudaErrors(hipDeviceSynchronize());

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) PRINTF("%s\n", hipGetErrorString(err));

//check for errors
    checkCudaErrors(
        hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipDeviceSynchronize());

    PRINTF("Code %i\n", h_info);
    if (0 != h_info) {
        ERR0("Error: Cholesky factorization failed\n");
    }
//calculate x = A\b
    checkCudaErrors(
        hipsolverDnDpotrs(handle, uplo, individuals, 1, 
            d_matrix, individuals, d_b,
             individuals, info));

    checkCudaErrors(hipDeviceSynchronize());
    

//copy  solution from device to vector on host
    hipMemcpy(vector, d_b, sizeof(double) * individuals, hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) PRINTF("Memcpy: %s\n", hipGetErrorString(err));

            
//free allocated memory
    checkCudaErrors(hipFree(info));
    checkCudaErrors(hipFree(buffer));
    checkCudaErrors(hipFree(d_matrix));
    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipsolverDnDestroy(handle));
    checkCudaErrors(hipStreamDestroy(stream));
};

__global__ void scalar_int_naive(int8_t* d_M, double* d_A, Uint n, Uint k, Uint n_threads, size_t pitch){
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    int i = -1, j = 0;
    Uint sum = 0;

  //  printf("Pitch: %d, Necessary: %d\n", (int) pitch, k * (int) sizeof(int8_t));
  //  for (int i = 0; i< pitch * n; i++ )  printf( (i%pitch)? "%d ": "\n%d ", d_M[i]);
    if(thread < n_threads){
        //This calculates the position the thread is supposed to compute in the result matrix A
        for (; thread >= 0; j = thread + (++i), thread -= (n- i));
        //Now i,j denotes the index of A_{i,j} 
     /*   for (int m = 0; m < k; m++) sum += (Uint) (d_M[ i * k + m] * d_M[ j * k + m]); 
        d_A[ j * n + i] = d_A[ i * n + j] = (double) sum;
*/
    }
}
void gpu_relmat_custom(Uint* M, double* A, Uint snps, Uint individuals){
/*
    Calculates the crossproduct of M with cublas and stores the result in A.
    Input:
        M: non-encoded matrix of dimension snps x indiv (k x n) storing genomic information
        A: pointer to result matrix
        snps: Number of snps
        individuals: number of individuals
    Output:
        A: matrix containing the type-casted result of M^T * M
    
    Note: cublas is fortran based and therefore assumes M is column-major. Therefore to calculate
        A we instruct cublasgemmex to calculate M * M^T and adjust its parameters.
        Furthermore, gemmex requires the matrix M to have a row number that is a multiple of four
        Therefore this function implements a zero-padding to add extra rows
*/

//Define auxiliary variables as needed for gemmex
    Uint n = individuals;
    Uint k = snps;
    Uint dim = n * k;
    hipError_t err;
//Start timing copy and calculation time
#ifdef DEBUG
    std::chrono::time_point<std::chrono::high_resolution_clock> timer_start;
    std::chrono::time_point<std::chrono::high_resolution_clock> timer_stop;
    timer_start = std::chrono::high_resolution_clock::now();
#endif
// allocate memory
    double *d_A;
    int8_t *h_M, *d_M;
    int32_t *h_C;
    size_t pitch_M, pitch_A;
    hipMallocPitch((void**)&d_M, &pitch_M, (size_t)( sizeof(int8_t) * k), (size_t) n);
    hipMallocPitch((void**)&d_A, &pitch_A,(size_t) (sizeof(double) * n),(size_t) n );
    hipHostMalloc((void **)&h_M, sizeof(int8_t) * dim);
    hipHostMalloc((void **)&h_C, sizeof(int32_t) * n * n);
    checkCudaErrors(err);

    KEY_type *KT = KEYT();
    int cores = KT->global_utils.basic.cores;

    
//Type-cast matrix M to int8 and store the result in page-locked memory
//Zero-pad matrix to get a row number that is a multiple of four
#ifdef DO_PARALLEL
#pragma omp parallel for num_threads(GreaterZero(cores))   
#endif
    for (int i = 0; i < n; i++){
        for (int j = 0; j < k; j++){
        h_M[j + i * k] = (int8_t) (j< k ?  M[j + i * k] : 0 );
        }
    }


//Copy int8 matrix to device
    size_t M_width = (size_t)( sizeof(int8_t) * k);
    PRINTF("Copying: Pitch %d and h_M %d\n", (int) pitch_M * k, sizeof(int8_t) * dim);
    hipMemcpy2D(d_M, pitch_M, h_M, M_width, M_width, n, hipMemcpyHostToDevice);  
    checkCudaErrors(err);
//Calculate the crossproduct and check for errros
    Uint n_threads = n* (n+1)/2 ;
    scalar_int_naive <<< 1,1 >>> (d_M, d_A, n, k, n_threads, pitch_M);

//    scalar_int_naive <<< (1 + ( n_threads - 1)/ THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (d_M, d_A, n, k, n_threads);
    // PRINTF("GemmEx failed.");
    hipDeviceSynchronize();


//copy result back to host
    hipMemcpy(A, d_A, sizeof(double) * n * n, hipMemcpyDeviceToHost);

//Free memory 
    hipFree(d_M);
    hipFree(d_A);
    hipHostFree(h_C);
    hipHostFree(h_M);
    hipDeviceSynchronize();

//Stop timer
#ifdef DEBUG
    timer_stop = std::chrono::high_resolution_clock::now();
    PRINTF("Time: %.3f s\n", ((float) std::chrono::duration_cast<std::chrono::microseconds>(timer_stop - timer_start).count())/1000000.0 );
#endif
} 



void gpu_relmat_cublas(Uint* M, double* A, Uint snps, Uint individuals){
    /*
        Calculates the crossproduct of M with cublas and stores the result in A.
        Input:
            M: non-encoded matrix of dimension snps x indiv (k x n) storing genomic information
            A: pointer to result matrix
            snps: Number of snps
            individuals: number of individuals
        Output:
            A: matrix containing the type-casted result of M^T * M
        
        Note: cublas is fortran based and therefore assumes M is column-major. Therefore to calculate
            A we instruct cublasgemmex to calculate M * M^T and adjust its parameters.
            Furthermore, gemmex requires the matrix M to have a row number that is a multiple of four
            Therefore this function implements a zero-padding to add extra rows
    */
    
    //Define auxiliary variables as needed for gemmex
        Uint n = individuals;
        Uint m = individuals;
        Uint k = snps;
    
    //Auxiliary padding variables for padding
        Uint k_pad_diff = (PADDIM - k % PADDIM) % PADDIM;
        Uint k_pad = k + k_pad_diff;
        Uint dim = m * k_pad;
    
    //Start timing copy and calculation time
    #ifdef DEBUG
        std::chrono::time_point<std::chrono::high_resolution_clock> timer_start;
        std::chrono::time_point<std::chrono::high_resolution_clock> timer_stop;
        timer_start = std::chrono::high_resolution_clock::now();
    #endif
    //Declare cublas variables and allocate memory
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        int8_t *d_M, *h_M;
        int32_t *d_C, *h_C;
        int32_t alpha = 1.f;
        int32_t beta = 0.f;
        hipMalloc(&d_M, sizeof(int8_t) * dim);
        hipMalloc(&d_C, sizeof(int32_t) * n * m );
        hipHostMalloc((void **)&h_M, sizeof(int8_t) * dim);
        hipHostMalloc((void **)&h_C, sizeof(int32_t) * n * m);
    
    
	KEY_type *KT = KEYT();
  int cores = KT->global_utils.basic.cores;

    //Type-cast matrix M to int8 and store the result in page-locked memory
    //Zero-pad matrix to get a row number that is a multiple of four
    #ifdef DO_PARALLEL
    #pragma omp parallel for num_threads(GreaterZero(cores))   
    #endif
        for (int i = 0; i < n; i++){
            for (int j = 0; j < k_pad; j++){
            h_M[j + i * k_pad] = (int8_t) (j< k ?  M[j + i * k] : 0 );
            }
        }
    
    
    //Copy int8 matrix to device
    hipMemcpy(d_M, h_M, sizeof(int8_t) * dim, hipMemcpyHostToDevice);  

    //Calculate the crossproduct and check for errros
        hipblasStatus_t stat = hipblasGemmEx(handle,
            HIPBLAS_OP_T,
            HIPBLAS_OP_N,
            n,
            m,
            k_pad,
            &alpha,
            d_M,
            HIP_R_8I,
            k_pad, // I have no idea why this doesnt need to be individuals, same below
            d_M,
            HIP_R_8I,
            k_pad,
            &beta,
            d_C,
            HIP_R_32I,
            n,
            HIP_R_32I, //HIPBLAS_COMPUTE_32I,
            HIPBLAS_GEMM_DEFAULT
            );
        
        if(stat) PRINTF("GemmEx failed.");
        hipDeviceSynchronize();
    
    
    //copy result back to host
        hipMemcpy(h_C, d_C, sizeof(int32_t) * n * m, hipMemcpyDeviceToHost);
    
    //Convert result to double and store it in output matrix A
    #ifdef DO_PARALLEL
    #pragma omp parallel for num_threads(GreaterZero(cores))   
    #endif
        for (int i = 0; i < n * m; i++) A[i] = (double) h_C[i];
    
    //Free memory 
        hipblasDestroy(handle);
        hipFree(d_M);
        hipFree(d_C);
        hipHostFree(h_C);
        hipHostFree(h_M);
    
    //Stop timer
    #ifdef DEBUG
        timer_stop = std::chrono::high_resolution_clock::now();
        PRINTF("Time: %.3f s\n", ((float) std::chrono::duration_cast<std::chrono::microseconds>(timer_stop - timer_start).count())/1000000.0 );
    #endif
    } 
/*
This is a possible alternative implementation in cublasLt. Might be faster?
    hipblasLtCreate(&handle);

    hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I);
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb));
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb));

    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8I, snps, in dividuals, snps);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8I, in dividuals, snps, snps);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32I, individuals, individuals, individuals);


    hipblasLtMatmulPreferenceCreate(&preference);
    cublasLtMatmulPreferenceInit(preference);

//Hopefully not needed:
    hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));

    hipblasLtMatmulAlgoGetHeuristic(handle, operationDesc, Adesc, Bdesc, Cdesc, Cdesc, preference, 1, &heuristicResult, &returnedResults);

    if (returnedResults == 0) {
       ERR1("Status %.50s", HIPBLAS_STATUS_NOT_SUPPORTED);
    }

    hipblasLtMatmul(handle,
        operationDesc,
        d_alpha,  // 1
        d_M,
        Adesc,
        d_M,
        Bdesc,
        d_beta,   // 0
        d_C,
        Cdesc,
        d_C,
        Cdesc,
        &heuristicResult.algo,
        workspace,
        workspaceSize,
        0);

    hipblasLtMatmulDescDestroy(operationDesc);
    hipblasLtDestroy(handle);
*/
