#include "hip/hip_runtime.h"
/*
 Authors 
 Alexander Freudenberg, alexander.freudenberg@stads.de

 Copyright (C) 2022-2023 Alexander Freudenberg

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
*/


#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string>
#include <inttypes.h>
#include <time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hipblas.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsolver.h>
#include <hipsparse.h>

#include "cuda_utils.h"

#define STR(x) XSTR(x)
#define XSTR(x) #x

void debug_info(const char *s, ...) {

  char *print_level_env = getenv("PRINT_LEVEL");
  if (print_level_env != NULL) {
    int print_level = atoi(print_level_env);

    va_list argptr;
    va_start(argptr, s);
    printf("\033[36m\t ");
    vprintf(s, argptr);
    printf(" \033[37m\n");
    va_end(argptr);
  }
}

int checkError(const char *func, int line, hipError_t err) {
  if (err != hipSuccess) {
    printf("Internal error in function %s at line %d: %s\n", func, line,
           hipGetErrorString(err));
    return 1;
  }
  return 0;
}

int checkError(const char *func, int line, hipblasStatus_t err) {
  if (err != HIPBLAS_STATUS_SUCCESS) {
    printf("Internal error in CUBLAS function %s at line %d: %s\n", func, line,
           cublasGetStatusString(err));
    return 1;
  }
  return 0;
}

int checkError(const char *func, int line, hipsparseStatus_t err) {
  if (err != HIPSPARSE_STATUS_SUCCESS) {
    printf("Error in call to cuSPARSE in function %s at line %d: %s\n", func, line,
           hipsparseGetErrorString(err));
    return 1;
  }
  return 0;
}

int checkError(const char *func, int line, hipsolverStatus_t err) {
  if (err != HIPSOLVER_STATUS_SUCCESS) {
    printf("Error in call to cuSOLVER in function %s at line %d: %d\n", func, line,
           err);
    return 1;
  }
  return 0;
}


int checkCuda(){
  //
  // Driver check
  // The following section checks if a compatible driver and runtime is
  // installed
  //
  hipError_t err;
  int driverVersion = 0, runtimeVersion = 0;

  err = hipDriverGetVersion(&driverVersion);
  if (checkError(__func__, __LINE__, err) != 0)
    return (1);
  if (driverVersion == 0) { // Check if there's a CUDA driver on the system
    printf("No CUDA driver detected.");
    return 1;
  }

  // Check if the CUDA runtime is compatible 
  hipRuntimeGetVersion(&runtimeVersion);
  int driverMajor = driverVersion / 1000,
      driverMinor = (driverVersion - driverMajor * 1000) / 10,
      runtimeMajor = runtimeVersion / 1000,
      runtimeMinor = (runtimeVersion - runtimeMajor * 1000) / 10;

  if (driverMajor <
      10) { // CUDA drivers below 10 don't have the required feature set
    printf("Your CUDA driver is of version %d.%d. CUDA driver versions below "
           "10.0 are not supported.\n",
           driverMajor, driverMinor);
    return 1;
  }
  if (driverVersion < runtimeVersion) { // Mismatches between runtime and driver
                                        // might lead to problems
    printf("This software has been compiled with CUDA version %d.%d, but "
           "your driver is of version %d.%d. If you run into errors, consider "
           "uprading your CUDA driver.\n",
           runtimeMajor, runtimeMinor, driverMajor, driverMinor);
  }

 return 0;
}

int checkDevMemory(size_t required_mem){
  //
  //  Device memory check
  // Checks if the device has enough RAM available

  hipError_t err;
  if (checkCuda() != 0)
      return 1;

  size_t free_mem = 0, total_mem = 0;
  err = hipMemGetInfo(&free_mem, &total_mem);
  if (checkError(__func__, __LINE__, err) != 0)
      return 1;

  if (free_mem < required_mem) {
      printf("CUDA Error: Not enough memory available. \nRequired %zu GB, free "
            "%zu GB, total on device %zu GB \n ",
            required_mem / size_t(pow(1024, 3)), free_mem / size_t(pow(1024, 3)),
            total_mem / size_t(pow(1024, 3)));
      return 1;
  }

  return 0;
}

int switchDevice(){
  //
  // Select GPU device
  // The following section switches the current context to the requested device
  // 

  hipError_t err;
  int device = 0;
  int device_available = 0;
  char *requested_device = getenv("CUDA_DEVICE");
  if (requested_device != NULL) {
    device = atoi(requested_device);
    printf("Environment variable CUDA_DEVICE is set to %s, switching to device %d.\n",
           requested_device, device);
  }
  else {
    printf("Environment variable CUDA_DEVICE is not set, using device 0.\n");
  }


  // Check if the requested device is available
  char *visible_devices =
      getenv("CUDA_VISIBLE_DEVICES"); // Get envirnoment variable
  if (visible_devices == NULL) {      // Check if variable is set
    printf("There are no visible CUDA devices. Check if there is a device "
           "available or set the CUDA_VISIBLE_DEVICES variable\n");
    return -1;
  }

  char *token = strtok(visible_devices, ","); // get first token
  while (token != NULL) {
    int val = atoi(token); // convert token to integer
    if (val == device) {   // break if device number is found in variable
      device_available = 1;
      break;
    }
    token = strtok(NULL, ","); // get next token
  }
  if (device_available == 0){ 
    printf("The requested device %d is not visible to the CUDA driver.\n",
           device);
    return -1;
  }

  // Switch to device
  err = hipSetDevice(device);
  if (checkError(__func__, __LINE__, err) != 0)
    return -1;

  return device;
}


