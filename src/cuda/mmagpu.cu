#include "hip/hip_runtime.h"
/*
 Alexander Freudenberg, alexander.freudenberg@stads.de

 Copyright (C) 2020-2023 Alexander Freudenberg

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Include order is important!!!
// Namespace conflicts if order is changed
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <inttypes.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <time.h>
#include <unistd.h>
#include <omp.h>

#include "cuda_utils.h"
#include "mmagpuIntern.h"

const int default_tile_size = 2048;

int gpuCrossprodIntern(char *snp_matrix, int snps,
                               int indiv, double *ans) {   
    /*
    xxx

    */
                              
    // Get number of threads
    hipError_t err         = hipSuccess,
                private_err = hipSuccess;
    hipStream_t stream;

    // Input data
    char *d_Z_block1, *d_Z_block2;
    // Buffer for output
    int *d_M, *h_M;

    bool verbose = get_print_level() >= 0;
    const char *env_num_threads = getenv("OMP_NUM_THREADS");
    int num_threads = 4;
    if (env_num_threads != NULL) {
        num_threads = atoi(env_num_threads);
    }
    if (verbose) {
        printf("Using %d OMP threads.\n", num_threads);
    }

    const long n_bytes_per_snp =
        (indiv - 1) / 4 + 1; // number of columns of Z if individuals
                             // are zero-padded to be a multiple of 4
    const long n_indiv_per_byte = 8L / 2L;

    // sanity checks
    // limit Tilesize to individuals
    const char *env_tile_size = getenv("TILE_SIZE");
    int mem_tile_size = default_tile_size;
    if (env_tile_size != NULL) {
        mem_tile_size = atoi(env_tile_size);
    }
    debug_info("Using tile size of %d.\n", mem_tile_size);

    mem_tile_size = min(indiv, mem_tile_size);

    if (checkCuda() != 0) {
        return 1;
    }
    int device = switchDevice();
    if (device == -1) {
        return 1;
    }

    // Calculate total memory requirements
    size_t required_mem = num_threads * (2 * n_bytes_per_snp * mem_tile_size +
                          mem_tile_size * mem_tile_size * sizeof(unsigned int));
    if (checkDevMemory(required_mem) != 0) {
        return 1;
    }

    int size_of_input = n_bytes_per_snp * mem_tile_size;
    int size_of_output = sizeof(int) * mem_tile_size * mem_tile_size;
    // Initialization of buffers: Calculate num_threads of tile matrix
    // multiplications in parallel and allocate the corresponding amount of
    // memory
    err = hipMalloc((void **)&d_Z_block1, num_threads * size_of_input);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    err = hipMalloc((void **)&d_Z_block2, num_threads * size_of_input);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    err = hipMalloc((void **)&d_M, num_threads * size_of_output);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    err = hipHostMalloc((void **)&h_M, num_threads * size_of_output);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);

    // initialization of cutlass gemm operators
    using ColumnMajor = cutlass::layout::ColumnMajor;
    using RowMajor = cutlass::layout::RowMajor;
    using TensorOp = cutlass::arch::OpClassTensorOp;
    using ElementA_ = cutlass::uint4b_t;
    using LayoutA_ = RowMajor;
    using ElementB_ = cutlass::uint4b_t;
    using LayoutB_ = ColumnMajor;
    using ElementC_ = int;
    using LayoutC_ = RowMajor;
    using ElementAccumulator_ = ElementC_;
    using OperatorClass_ = TensorOp;
    using ArchTag_ = cutlass::arch::Sm75;
    using ThreadblockShape_ =
        typename cutlass::gemm::device::DefaultGemmConfiguration<
            OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
            ElementAccumulator_>::ThreadblockShape;
    using WarpShape_ = typename cutlass::gemm::device::DefaultGemmConfiguration<
        OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
        ElementAccumulator_>::WarpShape;
    using InstructionShape_ =
        typename cutlass::gemm::device::DefaultGemmConfiguration<
            OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
            ElementAccumulator_>::InstructionShape;
    using EpilogueOutputOp_ =
        typename cutlass::gemm::device::DefaultGemmConfiguration<
            OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
            ElementAccumulator_>::EpilogueOutputOp;
    using ThreadblockSwizzle_ =
        typename cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;
    const int Stages = cutlass::gemm::device::DefaultGemmConfiguration<
        OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
        ElementAccumulator_>::kStages;
    const int AlignmentA = cutlass::gemm::device::DefaultGemmConfiguration<
        OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
        ElementAccumulator_>::kAlignmentA;
    const int AlignmentB = cutlass::gemm::device::DefaultGemmConfiguration<
        OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
        ElementAccumulator_>::kAlignmentB;
    const bool SplitKSerial = false;
    using Operator_ = typename cutlass::gemm::device::DefaultGemmConfiguration<
        OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
        ElementAccumulator_>::Operator;
    const bool IsBetaZero = false;
    using CutlassGemm = cutlass::gemm::device::Gemm<
        ElementA_, LayoutA_, ElementB_, LayoutB_, ElementC_, LayoutC_,
        ElementAccumulator_, OperatorClass_, ArchTag_, ThreadblockShape_,
        WarpShape_, InstructionShape_, EpilogueOutputOp_, ThreadblockSwizzle_,
        Stages, AlignmentA, AlignmentB, SplitKSerial, cutlass::arch::CustomOp,
        IsBetaZero>;

    // Define a CUTLASS GEMM type
    CutlassGemm gemm_operator;

    if (checkError(__func__, __LINE__, hipGetLastError()) != 0)
        return (1);

    // Main loop
    // Calculates matrix multiplications in parallel: Each thread in this loop sends
    // its data to a different stream on the device. The threads calculate
    // concurrently and send the output back to main memory. Memory copies are
    // asynchronous to take full advantage of the memory bandwidth.
    #ifdef DO_PARALLEL
    #pragma omp parallel for num_threads(num_threads) private(private_err,stream) schedule(dynamic)
    #endif
    for (long i = 0; i < indiv; i += mem_tile_size) {
        if(err != hipSuccess){
            continue;
        }

        int threadidx = omp_get_thread_num();

        private_err = hipStreamCreate(&stream);
        if (checkError(__func__, __LINE__, private_err) != 0) {
            err = private_err;
            continue;
        }

        hipStreamSynchronize(stream);

        cutlass::uint4b_t *d_tile1 =
            (cutlass::uint4b_t *)(d_Z_block1 +
                                  threadidx * mem_tile_size * n_bytes_per_snp);
        cutlass::uint4b_t *d_tile2 =
            (cutlass::uint4b_t *)(d_Z_block2 +
                                  threadidx * mem_tile_size * n_bytes_per_snp);

        char *x = snp_matrix + i * n_bytes_per_snp;

        int rows_remaining = indiv - i;
        int x_tile_size = min(mem_tile_size, rows_remaining);

        private_err = hipMemcpyAsync(d_tile1, x, x_tile_size * n_bytes_per_snp,
                            hipMemcpyHostToDevice, stream);

        hipStreamSynchronize(stream);
        if (checkError(__func__, __LINE__, private_err) != 0) {
            err = private_err;
            continue;
        }

        for (long j = i; j < indiv; j += mem_tile_size) {
            char *y = snp_matrix + j * n_bytes_per_snp;

            int columns_remaining = indiv - j;
            int y_tile_size = min(mem_tile_size, columns_remaining);

            private_err = hipMemcpyAsync(d_tile2, y, y_tile_size * n_bytes_per_snp,
                            hipMemcpyHostToDevice, stream);

            hipStreamSynchronize(stream);
            if (checkError(__func__, __LINE__, private_err) != 0) {
                err = private_err;
                continue;
            }


            // initialize gemm arguments
            CutlassGemm::Arguments args(
                {int(x_tile_size), int(y_tile_size),
                int(n_bytes_per_snp * n_indiv_per_byte)},
                {d_tile1, int(n_bytes_per_snp * n_indiv_per_byte)},
                {d_tile2, int(n_bytes_per_snp * n_indiv_per_byte)},
                {d_M + threadidx * mem_tile_size * mem_tile_size, int(y_tile_size)},
                {d_M + threadidx * mem_tile_size * mem_tile_size, int(y_tile_size)},
                {1, 0});
            hipStreamSynchronize(stream);

            // compute Multiplication
            cutlass::Status status;
        #pragma omp critical
            status = gemm_operator(args, nullptr, stream);

            hipStreamSynchronize(stream);
            if (checkError(__func__, __LINE__, (hipError_t) status) != 0) {
                err = (hipError_t) status;
                continue;
            }

            // Copy results back to host
            private_err = hipMemcpyAsync(h_M + threadidx * mem_tile_size * mem_tile_size,
                                    d_M + threadidx * mem_tile_size * mem_tile_size,
                                    mem_tile_size * mem_tile_size * sizeof(int),
                                    hipMemcpyDeviceToHost, stream);

            hipStreamSynchronize(stream);
            if (checkError(__func__, __LINE__, private_err) != 0) {
                err = private_err;
                continue;
            }

            for (long d1 = 0; d1 < x_tile_size; d1++) {
                for (long d2 = 0; d2 < y_tile_size; d2++) {
                  // Get result
                  int Mij = *(h_M + threadidx * mem_tile_size * mem_tile_size +
                              d2 + d1 * y_tile_size);
                  double *ans0 = ans + (i + d1), *ans1 = ans + (i + d1) * indiv;

                  ans0[(j + d2) * indiv] = (double)Mij;
                  ans1[j + d2] = (double)Mij;
                }
            }
        }

        hipStreamSynchronize(stream);
        private_err = hipGetLastError();
        if (checkError(__func__, __LINE__, private_err) != 0) {
            err = private_err;
            continue;
        }

        hipStreamDestroy(stream);
    }

    // Free memory
    hipFree(d_Z_block1);
    hipFree(d_Z_block2);
    hipFree(d_M);
    hipHostFree(h_M);

    return 0;
}


extern "C" {

void crossprod_mmagpu(char *snp_matrix, int snps, int indiv,
                      double *ans) {
    gpuCrossprodIntern(snp_matrix, snps, indiv, ans);
}

}
