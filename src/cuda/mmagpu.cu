#include "hip/hip_runtime.h"
/*
 Alexander Freudenberg, alexander.freudenberg@stads.de

 Copyright (C) 2020-2023 Alexander Freudenberg

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Include order is important!!!
// Namespace conflicts if order is changed
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <inttypes.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <time.h>
#include <unistd.h>

#include "cuda_utils.h"
#include "mmagpuIntern.h"

const int default_tile_size = 2048;

int gpuCrossprodIntern(unsigned int *CGM, size_t snps,
                               size_t individuals, double *ans) {   
    /*
    xxx

    */
                              
    // Get number of threads
    hipError_t err;
    // Input data
    cutlass::uint4b_t *d_Z_block1;
    cutlass::uint4b_t *d_Z_block2;
    // Buffer for output
    int32_t *d_M, *h_M;

    bool verbose = get_print_level() >= 0;
    const char *env_num_threads = getenv("OMP_NUM_THREADS");
    int num_threads = 4;
    if (env_num_threads != NULL) {
        num_threads = atoi(env_num_threads);
    }
    if (verbose) {
        printf("Using %d OMP threads.\n", num_threads);
    }

    long n_bytes_per_snp =
        (indiv - 1) / 4 + 1; // number of columns of Z if individuals
                             // are zero-padded to be a multiple of 4
    long n_indiv_per_byte = 8L / 2L;

    // sanity checks
    // limit Tilesize to individuals
    const char *env_tile_size = getenv("TILE_SIZE");
    int mem_tile_size = default_tile_size;
    if (env_tile_size != NULL) {
        mem_tile_size = atoi(env_tile_size);
    }
    debug_info("Using tile size of %d.\n", mem_tile_size);

    mem_tile_size = min(individuals, mem_tile_size);

    if (checkCuda() != 0) {
        return 1;
    }
    int device = switchDevice();
    if (device == -1) {
        return 1;
    }

    // Calculate total memory requirements
    size_t required_mem = num_threads * (2 * n_bytes_per_snp * mem_tile_size +
                          mem_tile_size * mem_tile_size * sizeof(unsigned int));
    if (checkDevMemory(required_mem) != 0) {
        return 1;
    }

    int size_of_input = n_bytes_per_snp * mem_tile_size;
    int size_of_output = sizeof(int32_t) * mem_tile_size * mem_tile_size;
    // Initialization of buffers: We calculate n_streams of tile matrix
    // multiplications in parallel and allocate the corresponding amount of memory
    err = hipMalloc((void **)&d_Z_block1, n_streams * size_of_input);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    err = hipMalloc((void **)&d_Z_block2, n_streams * size_of_input);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    err = hipMalloc((void **)&d_M, n_streams * size_of_output);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);
    err = hipHostMalloc((void **)&h_M, n_streams * size_of_output);
    if (checkError(__func__, __LINE__, err) != 0)
        return (1);

    // initialization of cutlass gemm operators
    using ColumnMajor = cutlass::layout::ColumnMajor;
    using RowMajor = cutlass::layout::RowMajor;
    using TensorOp = cutlass::arch::OpClassTensorOp;
    using ElementA_ = cutlass::uint4b_t;
    using LayoutA_ = RowMajor;
    using ElementB_ = cutlass::uint4b_t;
    using LayoutB_ = ColumnMajor;
    using ElementC_ = int32_t;
    using LayoutC_ = RowMajor;
    using ElementAccumulator_ = ElementC_;
    using OperatorClass_ = TensorOp;
    using ArchTag_ = cutlass::arch::Sm75;
    using ThreadblockShape_ =
        typename cutlass::gemm::device::DefaultGemmConfiguration<
            OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
            ElementAccumulator_>::ThreadblockShape;
    using WarpShape_ = typename cutlass::gemm::device::DefaultGemmConfiguration<
        OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
        ElementAccumulator_>::WarpShape;
    using InstructionShape_ =
        typename cutlass::gemm::device::DefaultGemmConfiguration<
            OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
            ElementAccumulator_>::InstructionShape;
    using EpilogueOutputOp_ =
        typename cutlass::gemm::device::DefaultGemmConfiguration<
            OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
            ElementAccumulator_>::EpilogueOutputOp;
    using ThreadblockSwizzle_ =
        typename cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;
    const int Stages = cutlass::gemm::device::DefaultGemmConfiguration<
        OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
        ElementAccumulator_>::kStages;
    const int AlignmentA = cutlass::gemm::device::DefaultGemmConfiguration<
        OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
        ElementAccumulator_>::kAlignmentA;
    const int AlignmentB = cutlass::gemm::device::DefaultGemmConfiguration<
        OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
        ElementAccumulator_>::kAlignmentB;
    const bool SplitKSerial = false;
    using Operator_ = typename cutlass::gemm::device::DefaultGemmConfiguration<
        OperatorClass_, ArchTag_, ElementA_, ElementB_, ElementC_,
        ElementAccumulator_>::Operator;
    const bool IsBetaZero = false;
    using CutlassGemm = cutlass::gemm::device::Gemm<
        ElementA_, LayoutA_, ElementB_, LayoutB_, ElementC_, LayoutC_,
        ElementAccumulator_, OperatorClass_, ArchTag_, ThreadblockShape_,
        WarpShape_, InstructionShape_, EpilogueOutputOp_, ThreadblockSwizzle_,
        Stages, AlignmentA, AlignmentB, SplitKSerial, cutlass::arch::CustomOp,
        IsBetaZero>;

    // Define a CUTLASS GEMM type
    CutlassGemm gemm_operator;

    if (checkError(__func__, __LINE__, hipGetLastError()) != 0)
    return (1);

    // Main loop
    // Calculates matrix multiplications in parallel: Each thread in this loop sends
    // its data to a different stream on the device. The threads calculate
    // concurrently and send the output back to main memory. Memory copies are
    // asynchronous to take full advantage of the memory bandwidth.
    #ifdef DO_PARALLEL
    #pragma omp parallel for num_threads(num_threads) schedule(dynamic)
    #endif
    for (int64_t i = 0; i < individuals; i += mem_tile_size) {
        int threadidx = omp_get_thread_num();
        hipStream_t stream;
        
         err = hipStreamCreate(&stream);
        hipStreamSynchronize(stream);

        // Pointer to the first element of current rows
        unsigned int *x = (CGM + i * IntsPerRow);
        cutlass::uint4b_t *d_tile1 =
            d_Z_block1 + threadidx * mem_tile_size * n_bytes_per_snp;
        cutlass::uint4b_t *d_tile2 =
            d_Z_block2 + threadidx * mem_tile_size * n_bytes_per_snp;

        // Number of rows in matrix
        size_t const rows_left = individuals - i;
        // Size x of current tile
        size_t const x_tile_size = min(mem_tile_size, rows_left);

        hipMemcpyAsync(d_tile1, x, x_tile_size * n_bytes_per_snp,
                        hipMemcpyHostToDevice, stream);

        hipStreamSynchronize(stream);
        err_check("Copy 1:");

        // Inner loop
        for (int64_t j = i; j < individuals; j += mem_tile_size) {

        // Same as above with y
        size_t const columns_left = individuals - j;
        size_t const y_tile_size = min(mem_tile_size, columns_left);
        unsigned int *y = (CGM + j * IntsPerRow);

        hipMemcpyAsync(d_tile2, y, y_tile_size * n_bytes_per_snp,
                        hipMemcpyHostToDevice, stream);
        err_check("Copy 2:");
        hipStreamSynchronize(stream);

        // initialize gemm arguments
        CutlassGemm::Arguments args(
            {int(x_tile_size), int(y_tile_size),
            int(n_bytes_per_snp * n_indiv_per_byte)},
            {d_tile1, int(n_bytes_per_snp * n_indiv_per_byte)},
            {d_tile2, int(n_bytes_per_snp * n_indiv_per_byte)},
            {d_M + threadidx * mem_tile_size * mem_tile_size, int(y_tile_size)},
            {d_M + threadidx * mem_tile_size * mem_tile_size, int(y_tile_size)}, {1, 0});
        hipStreamSynchronize(stream);

        // compute Multiplication
        cutlass::Status status;
    #pragma omp critical
        status = gemm_operator(args, nullptr, stream);

        hipStreamSynchronize(stream);
        err_check("Calculation:");

        // Copy results back to host
        hipMemcpyAsync(h_M + threadidx * mem_tile_size * mem_tile_size,
                        d_M + threadidx * mem_tile_size * mem_tile_size,
                        mem_tile_size * mem_tile_size * sizeof(int32_t),
                        hipMemcpyDeviceToHost, stream);
        err_check("Copying back:");

        hipStreamSynchronize(stream);


        err_check("Copy back:");

    // Loop over tile and store values in output matrix
    #ifdef DO_PARALLEL
    #pragma omp parallel for num_threads(num_threads) schedule(static)
    #endif
        for (int64_t di = 0; di < x_tile_size; ++di) {
            for (int64_t dj = 0; dj < y_tile_size; ++dj) {
            // Get result
            int32_t *Mij  = *(h_M + threadidx * mem_tile_size * mem_tile_size + dj + di * y_tile_size);
            double *ans0 = ans + (i + di),
                   *ans1 = ans + (i + di) * individuals;
            ans0[(j + dj) * individuals] = (double)Mij;
            ans1[j + dj] = (double)Mij;
            }
        }
        }
        hipStreamDestroy(stream);
    }

    // Free memory
    hipFree(d_Z_block1);
    hipFree(d_Z_block2);
    hipFree(d_M);
    hipHostFree(h_M);
}

static void crossprodIntern(Uint *CM, Uint snps, Uint individuals,
                            double *ans) {
    // tilse_size needs to stay the same: for smaller values we experience
    // undocumented calculation failures on the device
    const size_t tilesize = 2048;

    // Initialize host pointers and copy input data cuda managed memory
    Uint *h_CM;
    const size_t BytesPerIndiv = UnitsPerIndiv256(snps) * BytesPerUnit;
    hipHostMalloc((void **)&h_CM, individuals * BytesPerIndiv);
    MEMCOPY(h_CM, CM, individuals * BytesPerIndiv);

    gpuCrossprodIntern(h_CM, snps, individuals, ans, tilesize);
    hipHostFree(h_CM);
}

extern "C" {
void crossprod_mmagpu(Uint *CGM, Uint snps, Uint individuals, double *ans) {
crossprodIntern(CGM, snps, individuals, ans);
}
}
