

#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <inttypes.h>
#include <stdio.h>
#include <string>
#include <time.h>
#include <unistd.h>
#include "dgemm_compressed_cuda.h"

int main() {
  int device_count;
  hipGetDeviceCount(&device_count);
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasDestroy(handle);
  return 0;
}